
#include <hip/hip_runtime.h>


__constant__ float q = 1.60217646e-19;
__constant__ float m = 9.10938188e-31;
__constant__ float B0    = 1e-12;
__constant__ float alpha = 250000;

__global__ void kernel(float* x, float* y, float* z,
	float* vx, float* vy, float* vz, int count, float tau)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < count)
	{
		float r = sqrtf(x[i]*x[i]+y[i]*y[i]+z[i]*z[i]);

		float Bx = 0;
		float By = 0;
		float Bz = -B0 * expf(-r*r / alpha);

		float vx1 = vx[i]; float vy1 = vy[i]; float vz1 = vz[i];
		vx[i] = vx1 + tau * q * (vy1 * Bz - vz1 * By) / m;
		vy[i] = vy1 + tau * q * (vz1 * Bx - vx1 * Bz) / m;
		vz[i] = vz1 + tau * q * (vx1 * By - vy1 * Bx) / m;

		x[i] += vx[i] * tau;
		y[i] += vy[i] * tau;
		z[i] += vz[i] * tau;
	}
}

static float *d_x = NULL, *d_y = NULL, *d_z = NULL,
			 *d_vx = NULL, *d_vy = NULL, *d_vz = NULL;
static size_t oldcount = 0;

__host__ void process_particles(float* x, float* y, float* z,
	float* vx, float* vy, float*vz, size_t count, float tau)
{
	int size = count * sizeof(float);
	if(!d_x || oldcount != count)
	{
		hipFree(d_x);
		hipMalloc(&d_x, size);
		hipFree(d_y);
		hipMalloc(&d_y, size);
		hipFree(d_z);
		hipMalloc(&d_z, size);
		hipFree(d_vx);
		hipMalloc(&d_vx, size);
		hipFree(d_vy);
		hipMalloc(&d_vy, size);
		hipFree(d_vz);
		hipMalloc(&d_vz, size);
		oldcount = count;
	}
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
	hipMemcpy(d_z, z, size, hipMemcpyHostToDevice);
	hipMemcpy(d_vx, vx, size, hipMemcpyHostToDevice);
	hipMemcpy(d_vy, vy, size, hipMemcpyHostToDevice);
	hipMemcpy(d_vz, vz, size, hipMemcpyHostToDevice);

	kernel<<<count / 256 + 1, 256>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, count, tau);

	hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
	hipMemcpy(z, d_z, size, hipMemcpyDeviceToHost);
	hipMemcpy(vx, d_vx, size, hipMemcpyDeviceToHost);
	hipMemcpy(vy, d_vy, size, hipMemcpyDeviceToHost);
	hipMemcpy(vz, d_vz, size, hipMemcpyDeviceToHost);
}

